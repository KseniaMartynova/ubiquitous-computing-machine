#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <chrono>
#include <cmath>
#include <cstdlib>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSOLVER(func)                                                   \
{                                                                              \
    hipsolverStatus_t status = (func);                                          \
    if (status != HIPSOLVER_STATUS_SUCCESS) {                                   \
        printf("cuSOLVER API failed at line %d with error: %d\n",              \
               __LINE__, status);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("cuBLAS API failed at line %d with error: %d\n",                \
               __LINE__, status);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_size>" << std::endl;
        return EXIT_FAILURE;
    }

    int N = std::atoi(argv[1]);
    const int lda = N;  // Линейный размер матрицы

    // Выделение памяти на хосте
    double *h_A = (double*)malloc(lda * N * sizeof(double));
    double *h_A_inv = (double*)malloc(lda * N * sizeof(double));
    double *h_I = (double*)malloc(lda * N * sizeof(double));

    // Заполнение матрицы случайными значениями
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            h_A[i * lda + j] = (double)rand() / RAND_MAX;
        }
    }

    // Создание положительно определённой матрицы
    double *h_A_posdef = (double*)malloc(lda * N * sizeof(double));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            h_A_posdef[i * lda + j] = h_A[i * lda + j];
        }
    }
    for (int i = 0; i < N; i++) {
        h_A_posdef[i * lda + i] += N;
    }

    // Выделение памяти на устройстве
    double *d_A;
    CHECK_CUDA(hipMalloc((void**)&d_A, lda * N * sizeof(double)));

    // Копирование матрицы на устройство
    CHECK_CUDA(hipMemcpy(d_A, h_A_posdef, lda * N * sizeof(double), hipMemcpyHostToDevice));

    // Инициализация cuSOLVER
    hipsolverHandle_t cusolverH;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));

    // Выделение памяти для работы cuSOLVER
    int *devInfo;
    CHECK_CUDA(hipMalloc((void**)&devInfo, sizeof(int)));

    double *d_work;
    int lwork;
    CHECK_CUSOLVER(hipsolverDnDpotrf_bufferSize(cusolverH, HIPBLAS_FILL_MODE_UPPER, N, d_A, lda, &lwork));
    CHECK_CUDA(hipMalloc((void**)&d_work, lwork * sizeof(double)));

    // Измерение времени выполнения
    auto start = std::chrono::high_resolution_clock::now();

    // Выполнение разложения Холецкого
    CHECK_CUSOLVER(hipsolverDnDpotrf(cusolverH, HIPBLAS_FILL_MODE_UPPER, N, d_A, lda, d_work, lwork, devInfo));

    // Выполнение обращения матрицы
    CHECK_CUSOLVER(hipsolverDnDpotri(cusolverH, HIPBLAS_FILL_MODE_UPPER, N, d_A, lda, d_work, lwork, devInfo));

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;

    // Копирование результата на хост
    CHECK_CUDA(hipMemcpy(h_A_inv, d_A, lda * N * sizeof(double), hipMemcpyDeviceToHost));

    // Проверка корректности обращения
    double *d_I;
    CHECK_CUDA(hipMalloc((void**)&d_I, lda * N * sizeof(double)));

    // Инициализация cuBLAS
    hipblasHandle_t cublasH;
    CHECK_CUBLAS(hipblasCreate(&cublasH));

    // Умножение исходной матрицы на обратную
    double alpha = 1.0;
    double beta = 0.0;
    CHECK_CUBLAS(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, lda, d_A, lda, &beta, d_I, lda));

    // Копирование результата на хост
    CHECK_CUDA(hipMemcpy(h_I, d_I, lda * N * sizeof(double), hipMemcpyDeviceToHost));

    // Проверка на единичную матрицу
    bool correct = true;
    double tolerance = 1e-6;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            double expected = (i == j) ? 1.0 : 0.0;
            if (std::abs(h_I[i * lda + j] - expected) > tolerance) {
                correct = false;
                break;
            }
        }
        if (!correct) break;
    }

    if (correct) {
        std::cout << "Matrix inversion is correct." << std::endl;
    } else {
        std::cout << "Matrix inversion is incorrect." << std::endl;
    }

    // Вывод времени выполнения
    std::cout << "Time taken: " << duration.count() << " seconds" << std::endl;

    // Освобождение памяти
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(devInfo));
    CHECK_CUDA(hipFree(d_work));
    CHECK_CUDA(hipFree(d_I));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverH));
    CHECK_CUBLAS(hipblasDestroy(cublasH));

    free(h_A);
    free(h_A_inv);
    free(h_A_posdef);
    free(h_I);

    return 0;
}
